#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <algorithm>
#include <cassert>
#include <cstdint>
#include <cstring>

#include "AddSub.h"

namespace yabil
{

namespace bigint
{

constexpr int device_add_block_size = 64;

__global__ void add_with_carry(const bigint_base_t *a, const bigint_base_t *b, bigint_base_t *r, bigint_base_t *c)
{
    const int idx_c = blockDim.x * blockIdx.x + threadIdx.x;
    const int idx_r = idx_c * device_add_block_size;
    bigint_base_t carry = 0;

    for (int i = idx_r; i < idx_r + device_add_block_size; ++i)
    {
        const bigint_base_t tmp1 = a[i] + carry;
        carry = static_cast<bigint_base_t>(tmp1 < carry);
        const bigint_base_t tmp2 = (tmp1 + b[i]);
        carry += static_cast<bigint_base_t>(tmp2 < tmp1);
        r[i] = tmp2;
    }

    c[idx_c] = carry;
}

void add_arrays(const yabil::bigint::bigint_base_t *a, std::size_t a_size, const yabil::bigint::bigint_base_t *b,
                std::size_t b_size, yabil::bigint::bigint_base_t *r)
{
    assert(a_size >= b_size);

    constexpr int max_thread_count = 256;
    const auto max_common_size = static_cast<unsigned>(b_size);

    const auto blocks_count = max_common_size / max_thread_count;
    const auto results_output_size = blocks_count * max_thread_count;
    bigint_base_t carry = 0;

    if (blocks_count > 0)
    {
        const auto carry_output_size = results_output_size / device_add_block_size;
        bigint_base_t *carries = new bigint_base_t[carry_output_size];

        {
            bigint_base_t *a_device, *b_device, *results_device, *carries_device;
            hipMalloc(&a_device, results_output_size);
            hipMalloc(&b_device, results_output_size);
            hipMalloc(&results_device, results_output_size + 1);
            hipMalloc(&carries_device, carry_output_size);

            hipMemcpy(a_device, a, results_output_size, hipMemcpyHostToDevice);
            hipMemcpy(b_device, b, results_output_size, hipMemcpyHostToDevice);

            add_with_carry<<<blocks_count, max_thread_count>>>(a_device, b_device, results_device, carries_device);

            hipMemcpy(r, results_device, results_output_size, hipMemcpyDeviceToHost);
            hipMemcpy(carries, carries_device, carry_output_size, hipMemcpyDeviceToHost);

            hipFree(a_device);
            hipFree(b_device);
            hipFree(results_device);
            hipFree(carries_device);
        }

        for (unsigned i = 0; i < carry_output_size; ++i)
        {
            carry += carries[i];
            if (carry == 0)
            {
                continue;
            }

            for (unsigned j = (i + 1) * device_add_block_size; j < (i + 2) * device_add_block_size; ++j)
            {
                r[j] += 1;
                if (r[j] != 0)
                {
                    carry = 0;
                    break;
                }
            }
        }
        r[results_output_size] = carry;
        delete[] carries;
    }

    add_plain_arrays(&a[results_output_size], a_size - results_output_size, &b[results_output_size],
                     b_size - results_output_size, &r[results_output_size], carry);
}

void sub_arrays(const yabil::bigint::bigint_base_t *a, std::size_t a_size, const yabil::bigint::bigint_base_t *b,
                std::size_t b_size, yabil::bigint::bigint_base_t *r)
{
    sub_plain_arrays(a, a_size, b, b_size, r);
}
}  // namespace bigint

}  // namespace yabil
